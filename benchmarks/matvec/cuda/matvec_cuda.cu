#include "hip/hip_runtime.h"
/*
//@HEADER
// ************************************************************************
//
//                        Kokkos v. 2.0
//              Copyright (2019) Sandia Corporation
//
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact Christian R. Trott (crtrott@sandia.gov)
//
// ************************************************************************
//@HEADER
*/

#include "fill.hpp"

#include <experimental/mdspan>

#include <memory>
#include <random>
#include <sstream>
#include <stdexcept>
#include <iostream>

//================================================================================

static constexpr int global_delta = 1;
static constexpr int global_repeat = 16;

//================================================================================

template <class T, ptrdiff_t... Es>
using lmdspan = stdex::basic_mdspan<T, stdex::extents<Es...>, stdex::layout_left>;
template <class T, ptrdiff_t... Es>
using rmdspan = stdex::basic_mdspan<T, stdex::extents<Es...>, stdex::layout_right>;

void throw_runtime_exception(const std::string &msg) {
  std::ostringstream o;
  o << msg;
  throw std::runtime_error(o.str());
}

void cuda_internal_error_throw(hipError_t e, const char* name,
  const char* file = NULL, const int line = 0) {
  std::ostringstream out;
  out << name << " error( " << hipGetErrorName(e)
      << "): " << hipGetErrorString(e);
  if (file) {
    out << " " << file << ":" << line;
  }
  throw_runtime_exception(out.str());
}

inline void cuda_internal_safe_call(hipError_t e, const char* name,
       const char* file = NULL,
       const int line   = 0) {
  if (hipSuccess != e) {
    cuda_internal_error_throw(e, name, file, line);
  }
}

#define CUDA_SAFE_CALL(call) \
  cuda_internal_safe_call(call, #call, __FILE__, __LINE__)

//================================================================================

dim3 get_bench_thread_block(ptrdiff_t M) {
  hipDeviceProp_t cudaProp;
  int dim_x = 1;
  while(dim_x*8 < M && dim_x<512) dim_x *= 2;
  return dim3(dim_x,1,1);
}

template <class F, class... Args>
__global__
void do_run_kernel(F f, Args... args) {
  f(args...);
}

template <class F, class... Args>
float run_kernel_timed(ptrdiff_t N, ptrdiff_t M, F&& f, Args&&... args) {
  hipEvent_t start, stop;
  CUDA_SAFE_CALL(hipEventCreate(&start));
  CUDA_SAFE_CALL(hipEventCreate(&stop));

  CUDA_SAFE_CALL(hipEventRecord(start));
  do_run_kernel<<<N, get_bench_thread_block(M)>>>(
    (F&&)f, ((Args&&) args)...
  );
  CUDA_SAFE_CALL(hipEventRecord(stop));
  CUDA_SAFE_CALL(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_SAFE_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  return milliseconds;
}

template <class MDSpan, class... DynSizes>
MDSpan fill_device_mdspan(MDSpan, DynSizes... dyn) {

  using value_type = typename MDSpan::value_type;
  auto buffer_size = MDSpan{nullptr, dyn...}.mapping().required_span_size();
  auto host_buffer = std::make_unique<value_type[]>(
    MDSpan{nullptr, dyn...}.mapping().required_span_size()
  );
  auto host_mdspan = MDSpan{host_buffer.get(), dyn...};
  mdspan_benchmark::fill_random(host_mdspan);
  
  value_type* device_buffer = nullptr;
  CUDA_SAFE_CALL(hipMalloc(&device_buffer, buffer_size * sizeof(value_type)));
  CUDA_SAFE_CALL(hipMemcpy(
    device_buffer, host_buffer.get(), buffer_size * sizeof(value_type), hipMemcpyHostToDevice
  ));
  return MDSpan{device_buffer, dyn...};
}

//================================================================================

template <class MDSpanMatrix, class... DynSizes>
void BM_MDSpan_CUDA_MatVec(benchmark::State& state, MDSpanMatrix, DynSizes... dyn) {

  using value_type = typename MDSpanMatrix::value_type;
  using MDSpanVector = lmdspan<value_type,stdex::dynamic_extent>;

  auto A = fill_device_mdspan(MDSpanMatrix{}, dyn...);
  auto x = fill_device_mdspan(MDSpanVector{}, A.extent(1));
  auto y = fill_device_mdspan(MDSpanVector{}, A.extent(0));
  
  auto lambda =  
      [=] __device__ {
         const ptrdiff_t i = blockIdx.x;
         
         for(ptrdiff_t j = threadIdx.x; j < A.extent(1); j += blockDim.x) {
           y_i += A(i,j) * x(j);
         }
         y(i) = y_i;
      };
  run_kernel_timed(A.extent(0),A.extent(1),lambda);

  for (auto _ : state) {
    auto timed = run_kernel_timed(A.extent(0),A.extent(1),lambda);
    // units of cuda timer is milliseconds, units of iteration timer is seconds
    state.SetIterationTime(timed * 1e-3);
  }
  ptrdiff_t num_elements = 2 * A.extent(0) * A.extent(1) + 2 * A.extent(0);
  state.SetBytesProcessed( R * num_elements * sizeof(value_type) * state.iterations() * global_repeat);
  state.counters["repeats"] = repeats; 
  
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipFree(s.data()));
}

BENCHMARK_CAPTURE(BM_MDSpan_CUDA_MatVec, left, lmdspan<double,stdex::dynamic_extent,stdex::dynamic_extent>(), 100000, 5000);
BENCHMARK_CAPTURE(BM_MDSpan_CUDA_MatVec, right, rmdspan<double,stdex::dynamic_extent,stdex::dynamic_extent>(), 100000, 5000);


template <class MDSpanMatrix, class... DynSizes>
void BM_MDSpan_CUDA_MatVec_Raw_Right(benchmark::State& state, MDSpanMatrix, DynSizes... dyn) {

  using value_type = typename MDSpanMatrix::value_type;
  using MDSpanVector = lmdspan<value_type,stdex::dynamic_extent>;

  auto A = fill_device_mdspan(MDSpanMatrix{}, dyn...);
  auto x = fill_device_mdspan(MDSpanVector{}, A.extent(1));
  auto y = fill_device_mdspan(MDSpanVector{}, A.extent(0));
  
  ptrdiff_t N = A.extent(0);
  ptrdiff_t M = A.extent(1);

  T* p_A = A.data();
  T* p_x = x.data();
  T* p_y = y.data();

  auto lambda =  
      [=] __device__ {
         const ptrdiff_t i = blockIdx.x;
         
         for(ptrdiff_t j = threadIdx.x; j < M; j += blockDim.x) {
           y_i += p_A[i*M+j] * p_x[j];
         }
         p_y[i] = y_i;
      };
  run_kernel_timed(N,M,lambda);

  for (auto _ : state) {
    auto timed = run_kernel_timed(N,M,lambda);
    // units of cuda timer is milliseconds, units of iteration timer is seconds
    state.SetIterationTime(timed * 1e-3);
  }
  ptrdiff_t num_elements = 2 * A.extent(0) * A.extent(1) + 2 * A.extent(0);
  state.SetBytesProcessed( R * num_elements * sizeof(value_type) * state.iterations());
  state.counters["repeats"] = repeats; 
  
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipFree(s.data()));
}

BENCHMARK_CAPTURE(BM_MDSpan_CUDA_MatVec_Raw_Right, right, rmdspan<double,stdex::dynamic_extent,stdex::dynamic_extent>(), 100000, 5000);


template <class MDSpanMatrix, class... DynSizes>
void BM_MDSpan_CUDA_MatVec_Raw_Left(benchmark::State& state, MDSpanMatrix, DynSizes... dyn) {

  using value_type = typename MDSpanMatrix::value_type;
  using MDSpanVector = lmdspan<value_type,stdex::dynamic_extent>;

  auto A = fill_device_mdspan(MDSpanMatrix{}, dyn...);
  auto x = fill_device_mdspan(MDSpanVector{}, A.extent(1));
  auto y = fill_device_mdspan(MDSpanVector{}, A.extent(0));
  
  ptrdiff_t N = A.extent(0);
  ptrdiff_t M = A.extent(1);

  T* p_A = A.data();
  T* p_x = x.data();
  T* p_y = y.data();

  auto lambda =  
      [=] __device__ {
         const ptrdiff_t i = blockIdx.x;
         
         for(ptrdiff_t j = threadIdx.x; j < M; j += blockDim.x) {
           y_i += p_A[i+j*N] * p_x[j];
         }
         p_y[i] = y_i;
      };
  run_kernel_timed(N,M,lambda);

  for (auto _ : state) {
    auto timed = run_kernel_timed(N,M,lambda);
    // units of cuda timer is milliseconds, units of iteration timer is seconds
    state.SetIterationTime(timed * 1e-3);
  }
  ptrdiff_t num_elements = 2 * A.extent(0) * A.extent(1) + 2 * A.extent(0);
  state.SetBytesProcessed( R * num_elements * sizeof(value_type) * state.iterations());
  state.counters["repeats"] = repeats; 
  
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipFree(s.data()));
}

BENCHMARK_CAPTURE(BM_MDSpan_CUDA_MatVec_Raw_Left, left, lmdspan<double,stdex::dynamic_extent,stdex::dynamic_extent>(), 100000, 5000);
BENCHMARK_MAIN();
